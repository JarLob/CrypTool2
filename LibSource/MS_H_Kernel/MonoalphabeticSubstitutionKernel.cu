#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hiprand/hiprand.h> 
#include <hiprand/hiprand_kernel.h>


__global__ void kernelENG(long totalThreads, int* ciphertext, int textLength, int* runkey,
						double* quadgrams, double* cuda_out)
{
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	int index = x + y * blockDim.x;    //ThreadID. IMPORTANT: Build blocks (X*Y) with X=Y  !!! 2x2, 3x3 ...

	int plaintext[1000];	// 10000Must have constant Value, [textLength[0]] not possible. IMPORTANT: There wont be more then 10k Symbols loaded into kernel. Handeled in c# Code (HillclimbingAttacker).
	int i = index / 26;		// With i and j the Algorithm Computes the Chiddkey(K*). See the Modifyblock 
	int j = index % 26;
	

	int temp;
	double costvalue = 0;
	int threadKey[26];


	for (int k = 0; k < 26; k++)
	{
		threadKey[k] = runkey[k];
	}

	//K* = Modify K by swap position i and j
	temp = threadKey[i];
	threadKey[i] = threadKey[j];
	threadKey[j] = temp;

	//Plain = cipher, K*
	for (int k = 0; k < textLength; k++)
	{
		plaintext[k] = threadKey[ciphertext[k]];
	}

	//Costfunction
	int end = textLength -3;
	for (int k = 0; k < end; k++)
	{
		costvalue +=  quadgrams[plaintext[k] + (plaintext[k + 1] * 26) +
			(plaintext[k + 2]*26*26) +  (plaintext[k + 3]*26*26*26)];
	}

	//Output Return the Costvalue for each Thread
	for (int k = 0; k < totalThreads; k++)
	{
		cuda_out[index] = costvalue;
	}

}

__global__ void kernelGER(long totalThreads, int* ciphertext, int textLength, int* runkey,
						double* quadgrams, double* cuda_out)
{
	int x = blockIdx.x*blockDim.x+threadIdx.x;
	int y = blockIdx.y*blockDim.y+threadIdx.y;
	int index = x + y * blockDim.x;   //ThreadID. IMPORTANT: Build blocks (X*Y) with X=Y  !!! 2x2, 3x3 ...

	int plaintext[10000];	// Must have constant Value, [textLength[0]] not possible. IMPORTANT: There wont be more then 10k Symbols loaded into kernel. Handeled in c# Code (HillclimbingAttacker).
	int i = index / 30;		// With i and j the Algorithm Computes the Chiddkey(K*). See the Modifyblock 
	int j = index % 30;
	int temp;
	double costvalue = 0;
	int threadKey[30];

	for (int k = 0; k < 30; k++)
	{
		threadKey[k] = runkey[k];
	}

	//K* = Modify K by swap position i and j
	temp = threadKey[i];
	threadKey[i] = threadKey[j];
	threadKey[j] = temp;

	//Plain = cipher, K*
	for (int k = 0; k < textLength; k++)
	{
		plaintext[k] = threadKey[ciphertext[k]];
	}

	//Costfunction
	int end = textLength -3;	
	for (int k = 0; k < end; k++)
	{
		costvalue +=  quadgrams[plaintext[k] + (plaintext[k + 1] * 30) +
			(plaintext[k + 2]*30*30) +  (plaintext[k + 3]*30*30*30)];
	}

	//Output Return the Costvalue for each Thread
	for (int k = 0; k < totalThreads; k++)
	{
		cuda_out[index] = costvalue;
	}
}


int main()
{
    return 0;
}
